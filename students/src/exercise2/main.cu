#pragma warning( disable : 4244 ) 

#include <iostream>
#include <exercise2/Exercise2.h>


int main(int argc, const char**argv) 
{
    // find and start a device ...
    std::cout<<"Find the device ..." << std::endl;
    int bestDevice = findCudaDevice(argc, argv);
    checkCudaErrors( hipSetDevice( bestDevice ) );

    // launch the exercise 2
    Exercise2("Exercise 2").parseCommandLine(argc, argv).evaluate(true);

    // bye
    return 0;
}
