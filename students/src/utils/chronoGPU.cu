#include "chronoGPU.hpp"
#include <iostream>
#include "common.hpp"

using namespace std;

ChronoGPU::ChronoGPU() 
	: m_started( false ) {
	HANDLE_ERROR( hipEventCreate( &m_start ) );
	HANDLE_ERROR( hipEventCreate( &m_end ) );
}

ChronoGPU::~ChronoGPU() {
	if ( m_started ) {
		stop();
		std::cerr << "ChronoGPU::~ChronoGPU(): hrono wasn't turned off!" << std::endl; 
	}
	HANDLE_ERROR( hipEventDestroy( m_start ) );
	HANDLE_ERROR( hipEventDestroy( m_end ) );
}

void ChronoGPU::start() {
	if ( !m_started ) {
		HANDLE_ERROR( hipEventRecord( m_start, 0 ) );
		m_started = true;
	}
	else
		std::cerr << "ChronoGPU::start(): chrono is already started!" << std::endl;
}

void ChronoGPU::stop() {
	if ( m_started ) {
		HANDLE_ERROR( hipEventRecord( m_end, 0 ) );
		HANDLE_ERROR( hipEventSynchronize( m_end ) );
		m_started = false;
	}
	else
		std::cerr << "ChronoGPU::stop(): chrono wasn't started!" << std::endl;
}

float ChronoGPU::elapsedTime() {  
	float time = 0.f;
	if ( m_started ) {
		cerr << "ChronoCPU::elapsedTime(): chrono wasn't turned off!" << endl;
	}
	else {
		HANDLE_ERROR( hipEventElapsedTime( &time, m_start, m_end ) );
	}
	return time;
}