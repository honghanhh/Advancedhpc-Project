#include "hip/hip_runtime.h"
#pragma warning( disable : 4244 ) 

#include <iostream>
#include <exercise3/Exercise3.h>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/equal.h>


namespace {
        
}

// ==========================================================================================
void Exercise3::usage( const char*const prg ) {
    #ifdef WIN32
    const char*last_slash = strrchr(prg, '\\');
    #else
    const char*last_slash = strrchr(prg, '/');
    #endif
    std::cout << "Usage: " << (last_slash==nullptr ? prg : last_slash+1) 
        << " -i=<image.ppm> [-o=<image_output_basename.ppm>]"<< std::endl
        << "\twhere <image_input.ppm> is the input image," << std::endl
        << "\t<image_output_basename.ppm> is the basename of the output images." << std::endl
        << std::endl;
}

// ==========================================================================================
void Exercise3::usageAndExit( const char*const prg, const int code ) {
    usage(prg);
    exit( code );
}

// ==========================================================================================
void Exercise3::displayHelpIfNeeded(const int argc, const char**argv) 
{
    if( checkCmdLineFlag(argc, argv, "-h") || checkCmdLineFlag(argc, argv, "help") ) {
        usageAndExit(argv[0], EXIT_SUCCESS);
    }
}
Exercise3& Exercise3::parseCommandLine(const int argc, const char**argv) 
{
    if( checkCmdLineFlag(argc, argv, "i") ) {
        char *value;
        getCmdLineArgumentString(argc, argv, "i", &value);
        std::cout << "Input file is " << value << std::endl;
        inputFileName = std::string(value);
    }
    else
        usageAndExit(argv[0], -1); 
    if( checkCmdLineFlag(argc, argv, "o") ) {
        char*value;
        getCmdLineArgumentString(argc, argv, "o", &value);
        std::cout << "Output file is " << value << std::endl;
        outputFileName_rgb = std::string(value);
    }
    else {
        outputFileName_rgb = inputFileName;
    }
    reinterpret_cast<StudentWork3*>(student)->parseCommandLine(argc, argv);     
    return *this;
}

void Exercise3::loadImage() 
{
    input = new PPMBitmap(inputFileName.c_str());
    const unsigned size = input->getWidth()*input->getHeight();
    uchar3*const ptr = reinterpret_cast<uchar3*>( input->getPtr() );
    thrust::host_vector<uchar3> h_RGB( ptr, ptr+size);
    d_RGB_in = h_RGB;
    d_RGB_out.resize( d_RGB_in.size());
}

void Exercise3::run(const bool verbose) {    
    if( verbose )
        std::cout << std::endl << "Convert the image, build the histogram" << std::endl;
    // build a host vector containing the reference
    loadImage();
    ChronoGPU chr;
    StudentWork2& worker = *reinterpret_cast<StudentWork2*>(student);
    chr.start();
    worker.justDoIt( d_RGB_in, d_RGB_out, input->getWidth(), input->getHeight() );
    chr.stop();
    if( verbose )
        std::cout << "\t-> Student's Work Done in " << chr.elapsedTime() << " ms" << std::endl;
    saveImage(outputFileName_rgb.c_str(), d_RGB_out, input->getWidth(), input->getHeight());
}

bool Exercise3::check() {
    return true;
}



void Exercise3::saveImage(
    const char*filename, 
    const thrust::host_vector<uchar3>&h_image, 
    const unsigned width, 
    const unsigned height
) {
    PPMBitmap output(input->getWidth(), input->getHeight());
    thrust::copy(h_image.begin(), h_image.end(), reinterpret_cast<uchar3*>(output.getPtr()));
    output.saveTo(filename);
    std::cout << "Image saved to " << filename << std::endl;
}
