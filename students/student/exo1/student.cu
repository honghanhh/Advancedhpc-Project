#include "hip/hip_runtime.h"
#include "student.h"

namespace {
	
    template<typename T>
    __device__ inline T max(const T&a, const T&b) { 
        return a<b ? b : a; 
    }

	struct RGB2VFunctor : public thrust::unary_function<uchar3,uchar>
    {
        __device__
        uchar operator() (const uchar3& RGB) {
            return max(RGB.x, max(RGB.y, RGB.z)); // return the Value, i.e. the max
        }
    };
    
    struct FilterFunctor : public thrust::binary_function<const uchar3,const uchar,uchar3>
    {
        __device__
        uchar3 operator() (const uchar3&u_rgb, const uchar V) 
        {
            const float3 RGB = make_float3( float(u_rgb.x), float(u_rgb.y), float(u_rgb.z));
            const float d = fmaxf(RGB.x, fmaxf(RGB.y, RGB.z)); // old value
            const float N = d > 0.f ? float(V) / d : 0.f; // ratio
            const float R = fminf(RGB.x * N, 255.f);
            const float G = fminf(RGB.y * N, 255.f);
            const float B = fminf(RGB.z * N, 255.f);
            return make_uchar3(R, G, B); // modify the value of a given pixel
        }
	};
	
	// first pixel, fill all the shared memory with its neighbours
    __device__ void fill_shared_memory(
        const uchar*const d_V, 
        const int py,
        const unsigned width, const unsigned height, 
        const unsigned filter_size
    ) {
        extern __shared__ int s_Histo[];
        const int px = blockIdx.x;
        // we have exactly 256 threads
        s_Histo[threadIdx.x] = 0u;
        __syncthreads();
        const int startX = px - (filter_size>>1);
        const int startY = py - (filter_size>>1);
		for(unsigned tid=threadIdx.x; tid < filter_size*filter_size; tid+=blockDim.x )
        {
			// TODO: histogram with all neighbors
        }
        __syncthreads();
    }
    
    __device__ void update_histo(
        const uchar*const d_V, 
        const int py,
        const unsigned width, const unsigned height, 
        const unsigned filter_size
    ) {
        // need to remove the top line, and to add the bottom one
        extern __shared__ int s_Histo[];
        const int px = blockIdx.x;
        const int startX = px - (filter_size>>1);
        const int startY = py - (filter_size>>1);
        for(unsigned tid=threadIdx.x; tid < filter_size; tid+=blockDim.x )
        {
            // TODO: modify histogram, remove old top line, add new bottom one
        }
        __syncthreads();
    }

    __device__ void scan(const int py) 
    {
        extern __shared__ int s_mem[];
        const int *const s_Histo = &s_mem[0];
        volatile int *const s_scan = &s_mem[256];

        // 256 threads ...
        s_scan[threadIdx.x] = s_Histo[threadIdx.x];
        __syncthreads();
		// TODO: a scan into the current block (using shared memory)
    }

    __device__ void apply_filter(        
        const uchar*const d_V,
        uchar*const d_V_median,
        const int py,
        const unsigned width,
        const unsigned limit
    ) {
        extern __shared__ int s_mem[];
        const int *const s_cdf = &s_mem[256];
        // after scan, the histo is a CDF (cumulative distribution function)
        // then only only thread will succeed the following test ;-)
		// TODO
	}


#define CHECK
#ifdef CHECK
    __device__ void check_scan() 
    {
        extern __shared__ int s_mem[];
        const int *const s_scan = &s_mem[256];
        if( threadIdx.x>0 && s_scan[threadIdx.x-1]>s_scan[threadIdx.x] )
            printf("[%d/%d] bad values: %d\n", blockIdx.x, threadIdx.x, s_scan[threadIdx.x]);
    }
#endif

    __global__ void filter(
        const uchar*const d_V, 
        uchar*const d_V_median, 
		const unsigned width, 
		const unsigned height, 
        const unsigned filter_size
    ) {
        ::fill_shared_memory(d_V, 0, width, height, filter_size);
        // first pixel is specific (no maj): just scan and then apply filter
		::scan(0); 
#ifdef CHECK
        ::check_scan();
#endif
        ::apply_filter(d_V, d_V_median, 0, width, filter_size*filter_size/2);
        // others came after the first one, only updating the histo
        for(int py=1; py<height; ++py) 
        {
            // maj histo
            ::update_histo(d_V, py, width, height, filter_size);
            // scan
            ::scan(py); 
#ifdef CHECK
            ::check_scan();
#endif
            // apply
			::apply_filter(d_V, d_V_median, py, width, filter_size*filter_size/2);
        }
    }
}

bool StudentWork1::isImplemented() const {
	return false;
}


void StudentWork1::rgb2h(
	const thrust::device_vector<uchar3>&rgb,
	thrust::device_vector<uchar>&V
)
{
    thrust::transform(
        rgb.begin(),
        rgb.end(),
		V.begin(),
		::RGB2VFunctor()
    );
}

void StudentWork1::median(
	const thrust::device_vector<uchar> &d_V,
	thrust::device_vector<uchar> &d_V_median,
	const unsigned width,
	const unsigned height,
	const unsigned filter_size
) {    
	dim3 threads(256);    
	if( d_V.size() != width * height ) 
		std::cout << "Problem with the size of d_V" << std::endl;
	if( d_V_median.size() != width * height ) 
		std::cout << "Problem with the size of d_V_median" << std::endl;
    uchar const*const V = d_V.begin().base().get();
    uchar *const F = d_V_median.begin().base().get();
	dim3 blocks(width); 
	::filter<<<blocks, threads, sizeof(int)*512>>>(V, F, width, height, filter_size);
	std::cout << "do the copy" << std::endl;
}


void StudentWork1::apply_filter(
	const thrust::device_vector<uchar3>&RGB_old,
	const thrust::device_vector<uchar>&V_new,
	thrust::device_vector<uchar3>&RGB_new
) 
{
    thrust::transform(
        RGB_old.begin(), RGB_old.end(),
        V_new.begin(),
        RGB_new.begin(),
        ::FilterFunctor()
    );
}